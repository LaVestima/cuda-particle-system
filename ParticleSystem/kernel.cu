#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <cmath>
#include <ctime>
#include <Windows.h>

using namespace std;

const int epochs = 700;
const int parameterCount = 5;
const bool display = true;

inline hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess) {
		cout << "CUDA Runtime Error: " << hipGetErrorString(result) << endl;
	}

	return result;
}

__global__ void kernel(float *particles, int particleCount)
{
	extern __shared__ float sharedParticles[];

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	float x = particles[parameterCount * index];
	float y = particles[parameterCount * index + 1];
	float mass = particles[parameterCount * index + 2];
	float vx = particles[parameterCount * index + 3];
	float vy = particles[parameterCount * index + 4];

	sharedParticles[parameterCount * index] = x;
	sharedParticles[parameterCount * index + 1] = y;
	sharedParticles[parameterCount * index + 2] = mass;
	sharedParticles[parameterCount * index + 3] = vx;
	sharedParticles[parameterCount * index + 4] = vy;
	__syncthreads();

	float G = 0.05;
	float dt = 0.2;
	float e = 0.5;
	
	for (int i = 0; i < epochs; i++) {
		float fx = 0;
		float fy = 0;

		for (int j = 0; j < particleCount; j++) {
			if (j != index) {
				float jx = sharedParticles[parameterCount * j];
				float jy = sharedParticles[parameterCount * j + 1];
				float jmass = sharedParticles[parameterCount * j + 2];

				if (x != jx) {
					fx = fx + (G * mass * jmass * -(x - jx)) / pow(pow(abs(x - jx), 2) + e*e, 3 / 2);
				}
				if (y != jy) {
					fy = fy + (G * mass * jmass * -(y - jy)) / pow(pow(abs(y - jy), 2) + e*e, 3 / 2);
				}
			}
		}

		vx = vx + fx / mass * dt;
		vy = vy + fy / mass * dt;

		x = x + vx * dt;
		y = y + vy * dt;

		if (display) {
			//printf("%d\t%d\t%f\t%f\t%f\t%f\t%f\t%f\n", i, index, x, y, fx, fy, vx, vy);
			printf("%f\t%f\n", x, y);
		}

		__syncthreads();

		sharedParticles[parameterCount * index] = x;
		sharedParticles[parameterCount * index + 1] = y;
		sharedParticles[parameterCount * index + 2] = mass;
		sharedParticles[parameterCount * index + 3] = vx;
		sharedParticles[parameterCount * index + 4] = vy;
		__syncthreads();
	}
}

int main()
{
	srand((int)time(nullptr));

	bool random = true;

	const int particlesSize = 3;
	float particles[parameterCount * particlesSize];

	if (random) {
		for (int r = 0; r < particlesSize; r++) {
			particles[parameterCount * r] = (float)((rand() % 10000) / 100.0);
			particles[parameterCount * r + 1] = (float)((rand() % 10000) / 100.0);
			particles[parameterCount * r + 2] = (float)((rand() % 10000) / 100.0);
			particles[parameterCount * r + 3] = (float)((rand() % 200 - 100) / 100.0);
			particles[parameterCount * r + 4] = (float)((rand() % 200 - 100) / 100.0);
		}
	}
	else {
		// syntax: {x, y, mass}
		float newParticles[] = {
			10, 25, 140, 0, -0.5,
			25, 45, 10, 0, 2,
			-35, 35, 90.3, 0, 1.1
			//50, 70, 14.2,
			//35, 40, 35.4
		};
		copy(newParticles, newParticles + sizeof(newParticles)/sizeof(float), particles);
	}

	if (display) {
		cout << "index\tx\t\ty\t\tmass\tvx\tvy" << endl;
	}

	if (display) {
		for (int i = 0; i < particlesSize; i++) {
			printf("%d\t%f\t%f\t%f\t%f\t%f\n", 
				i, 
				particles[parameterCount * i], 
				particles[parameterCount * i + 1], 
				particles[parameterCount * i + 2], 
				particles[parameterCount * i + 3], 
				particles[parameterCount * i + 4]
			);
		}
	}

	if (display) {
		cout << endl << "epoch\tindex\tx\t\ty\t\tfx\t\tfy\t\tvx\t\tvy" << endl;
	}

	float endParticles[2][3];

	double tt, tu;
	LARGE_INTEGER ti, to, tb, te, tk, tm, tf;
	QueryPerformanceFrequency(&tf);

	dim3 threadsPerBlock(particlesSize);
	dim3 blocksPerGrid(1);

	float *cudaParticles;

	QueryPerformanceCounter(&tb);

	checkCuda(hipMalloc((void**)&cudaParticles, sizeof(particles)));
	checkCuda(hipMemcpy(cudaParticles, particles, sizeof(particles), hipMemcpyHostToDevice));

	QueryPerformanceCounter(&tk);
	kernel << <blocksPerGrid, threadsPerBlock, particlesSize*parameterCount*4 >> >(cudaParticles, particlesSize);
	QueryPerformanceCounter(&tm);

	checkCuda(hipMemcpy(endParticles, cudaParticles, sizeof(particles), hipMemcpyDeviceToHost));
	hipFree(cudaParticles);

	QueryPerformanceCounter(&te);
	tu = 1000.0*(double(tm.QuadPart - tk.QuadPart)) / double(tf.QuadPart);
	cout << "\n\nGPU time: " << tu << " ms\n";
	tt = 1000.0*(double(te.QuadPart - tb.QuadPart)) / double(tf.QuadPart);
	cout << "\n\nGPU time with memory operations: " << tt << " ms\n";

	checkCuda(hipDeviceReset());

	cout << "DONE";
	cin.ignore();

    return 0;
}
